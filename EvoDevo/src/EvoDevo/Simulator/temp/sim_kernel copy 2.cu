#include "hip/hip_runtime.h"
#include "vec_math.cuh"
#include "material.h"
#include <math.h>
#include <assert.h>

#define EPS (float) 1e-12
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
namespace EvoDevo {

struct SimOptions {
	float dt;
	uint massesPerBlock;
	uint springsPerBlock;
	uint facesPerBlock;
	uint cellsPerBlock;
	uint boundaryMassesPerBlock;
	uint maxMasses;
	uint maxSprings;
	uint maxFaces;
	uint maxCells;
	uint compositeCount;
	short shiftskip;
	float drag;
	float damping;
	float relaxation;
	float s;
};

struct DeviceData {
	// MASS DATA
	float    *dPos, *dNewPos, *dVel;
	uint32_t *dMassMatEncodings;

	// SPRING DATA
	ushort	 *dPairs;
	uint32_t *dSpringMatEncodings;
	uint8_t  *dSpringMatIds;
	float	 *dLbars;
	uint     *dSpringIDs;
	float	 *dSpringStresses;
	
	// SPRING DEVO DATA
	ushort   *dRandomPairs;
	uint     *dSpringIDs_Sorted;
	float	 *dSpringStresses_Sorted;

	// FACE DATA
	ushort	 *dFaces;
	
	// CELL DATA
	ushort	 *dCells;
	float	 *dVbars;
	float	 *dMats;
	float	 *dCellStresses;
};

__constant__ float4 compositeMats_id[COMPOSITE_COUNT];
__constant__ SimOptions cSimOpt;

void setCompositeMats_id(float* compositeMats, uint count) {
	hipMemcpyToSymbol(HIP_SYMBOL(compositeMats_id), compositeMats, sizeof(float)*4*count);
}

void setSimOpts(SimOptions opt) {
	hipMemcpyToSymbol(HIP_SYMBOL(cSimOpt), &opt, sizeof(SimOptions));
}

__global__ inline
void surfaceDragForce(float4 *__restrict__ pos, float4 *__restrict__ newPos,
                 float4 *__restrict__ vel, ushort4 *__restrict__ faces) {
	extern __shared__ float3 s[];
	float3  *s_pos = s;
	float3  *s_vel = (float3*) &s_pos[cSimOpt.boundaryMassesPerBlock];
	float3  *s_force = (float3*) &s_vel[cSimOpt.boundaryMassesPerBlock];
	
	uint massOffset   = blockIdx.x * cSimOpt.massesPerBlock;
	uint faceOffset   = blockIdx.x * cSimOpt.facesPerBlock;
	uint i;

	int tid    = threadIdx.x;
	int stride = blockDim.x;

	// Initialize and compute environment forces
	float4 pos4, vel4;
	for(i = tid; i < cSimOpt.boundaryMassesPerBlock && (i+massOffset) < cSimOpt.maxMasses; i+=stride) {
		pos4 = __ldg(&pos[i+massOffset]);
		vel4 = __ldg(&vel[i+massOffset]);
		s_pos[i] = {pos4.x,pos4.y,pos4.z};
		s_vel[i] = {vel4.x,vel4.y,vel4.z};
		s_force[i] = {0.0f, 0.0f, 0.0f};
	}
	
	float rho = cSimOpt.drag, area;
	ushort4 face;
	float3  x0, x1, x2,
	        v0, v1, v2,
			v, normal, force;
	for(i = tid; i < cSimOpt.facesPerBlock && (i+faceOffset) < cSimOpt.maxFaces; i+=stride) {
		// Drag Force: 0.5*rho*A*((Cd - Cl)*dot(v,n)*v + Cl*dot(v,v)*n)
		face = __ldg(&faces[i+faceOffset]);
		if(face.x == face.y || face.x == face.z || face.y == face.z)
			continue;

		x0 = s_pos[face.x];
		x1 = s_pos[face.y];
		x2 = s_pos[face.z];
		v0 = s_vel[face.x];
		v1 = s_vel[face.y];
		v2 = s_vel[face.z];

		
		v = (v0 + v1 + v2) / 3.0f;
		normal = cross((x1 - x0), (x2-x0));
		area = norm3df(normal.x,normal.y,normal.z);
		normal = normal / (area + EPS);
		normal = dot(normal, v) > 0.0f ? normal : -normal;
		force = -0.5f*rho*area*(0.8*dot(v,normal)*v + 0.2*dot(v,v)*normal);
		force = force / 3.0f; // allocate forces evenly amongst masses
		
		atomicAdd(&(s_force[face.x].x), force.x);
		atomicAdd(&(s_force[face.x].y), force.y);
		atomicAdd(&(s_force[face.x].z), force.z);

		atomicAdd(&(s_force[face.y].x), force.x);
		atomicAdd(&(s_force[face.y].y), force.y);
		atomicAdd(&(s_force[face.y].z), force.z);

		atomicAdd(&(s_force[face.z].x), force.x);
		atomicAdd(&(s_force[face.z].y), force.y);
		atomicAdd(&(s_force[face.z].z), force.z);
	}

	for(i = tid; i < cSimOpt.boundaryMassesPerBlock && (i+massOffset) < cSimOpt.maxMasses; i+=stride) {
		force = s_force[i];
		vel[i+massOffset].x += force.x*cSimOpt.dt;
		vel[i+massOffset].y += force.y*cSimOpt.dt;
		vel[i+massOffset].z += force.z*cSimOpt.dt;
	}
}

__global__
void pointDragForce(float4 *__restrict__ pos, float4 *__restrict__ newPos,
                 float4 *__restrict__ vel) {
	int stride = blockDim.x * gridDim.x;
	float4 velocity, pos4;

	float3 force;
	float  mag_vel;
	for(uint i = blockIdx.x * blockDim.x + threadIdx.x;
		i < cSimOpt.maxMasses; i+=stride) {
		velocity = __ldg(&vel[i]);
		mag_vel = norm3df(velocity.x,velocity.y,velocity.z);

		//Force due to drag = - (1/2 * rho * |v|^2 * A * Cd) * v / |v| (Assume A and Cd are 1)
		force.x = -0.5*cSimOpt.drag*mag_vel*velocity.x;
		force.y = -0.5*cSimOpt.drag*mag_vel*velocity.y;
		force.z = -0.5*cSimOpt.drag*mag_vel*velocity.z;

		pos4 = __ldg(&pos[i]);
		newPos[i].x = pos4.x + velocity.x*cSimOpt.dt + force.x*cSimOpt.dt*cSimOpt.dt;
		newPos[i].y = pos4.y + velocity.y*cSimOpt.dt + force.y*cSimOpt.dt*cSimOpt.dt;
		newPos[i].z = pos4.z + velocity.z*cSimOpt.dt + force.z*cSimOpt.dt*cSimOpt.dt;
	}
}


// __global__ inline
// void preSolve(float4 *__restrict__ pos, float4 *__restrict__ newPos,
//                  float4 *__restrict__ vel) {
// 	int stride = blockDim.x * gridDim.x;
// 	float4 velocity;

// 	for(uint i = blockIdx.x * blockDim.x + threadIdx.x;
// 		i < cSimOpt.maxMasses; i+=stride) {
		
// 		velocity = __ldg(&vel[i]);
// 		newPos[i] = __ldg(&pos[i]) + velocity*cSimOpt.dt;
// 	}
// }


/*
	Exended Positon Based Dynamics
	Computes lagrangian (force) for each distance constraint (spring)

	mat: float4 describing spring material
		x - k		stiffness
		y - dL0 	maximum percent change
		z - omega	frequency of oscillation
		w - phi		phase
*/
__global__ 
void solveDistance(
	float4 *__restrict__ pos,
	float4 *__restrict__ newPos,
	float4 *__restrict__ vel,
	ushort2 *__restrict__ pairs, 
	float * __restrict__ stresses,
	uint8_t *__restrict__ matIds,
	float *__restrict__ Lbars,
	float time, uint step, bool integrateForce
	)
{
	extern __shared__ float3 s[];
	float3  *s_pos = s;
	float3  *s_dp = (float3*) &s_pos[cSimOpt.massesPerBlock];
	
	uint massOffset   = blockIdx.x * cSimOpt.massesPerBlock;
	uint springOffset = blockIdx.x * cSimOpt.springsPerBlock;
	uint i;

	int tid    = threadIdx.x;
	int stride = blockDim.x;
	
	// Initialize and compute environment forces
	float4 pos4, vel4;
	for(i = tid; i < cSimOpt.massesPerBlock && (i+massOffset) < cSimOpt.maxMasses; i+=stride) {
		pos4 = __ldg(&newPos[i+massOffset]);
		vel4 = __ldg(&vel[i+massOffset]);
		// newPos[i] = pos4 + velocity*cSimOpt.dt;
		s_pos[i] = {
			pos4.x + vel4.x*cSimOpt.dt,
			pos4.y + vel4.y*cSimOpt.dt,
			pos4.z + vel4.z*cSimOpt.dt
		};
	}
	
	for(i = tid; i < cSimOpt.massesPerBlock && (i+massOffset) < cSimOpt.maxMasses; i+=stride) {
		s_dp[i] = {0.0f, 0.0f, 0.0f};
	}

	__syncthreads();

	float4	mat;
	uint8_t matId;

	float3	pos0, pos1;
	float	Lbar,
			C, alpha,
			lambda;
	ushort  v0, v1;
	ushort2	pair;

	float3	distance, n;

	float	relative_change,
			rest_length,
			d, K;
	float3	dp;
	
	for(i = tid; i < cSimOpt.springsPerBlock && (i+springOffset) < cSimOpt.maxSprings; i+=stride) {
		matId = __ldg(&matIds[i+springOffset]);
		if(matId == materials::air.id) continue;

		pair = __ldg(&pairs[i+springOffset]);
		Lbar = __ldg(&Lbars[i+springOffset]);
		v0 = pair.x; v1 = pair.y;
		pos0 = s_pos[v0];
		pos1 = s_pos[v1];

		mat = compositeMats_id[ matId ];
		alpha = 1.0f/ mat.x / cSimOpt.dt / cSimOpt.dt;
		// rest_length = mean_length * (1 + relative_change);
		relative_change = mat.y * sinf(mat.z*time+mat.w);
		rest_length = __fmaf_rn(Lbar, relative_change, Lbar);
		
		K = 2.0f + alpha;
		distance = pos0-pos1;
		d = l2norm(distance);
		n = distance / (d + EPS);
		
		C = d-rest_length;
		lambda = -(C) / (K);
		dp = lambda * n;

		if(integrateForce) stresses[i+springOffset] += lambda / Lbar;

		atomicAdd(&(s_dp[v0].x), dp.x);
		atomicAdd(&(s_dp[v0].y), dp.y);
		atomicAdd(&(s_dp[v0].z), dp.z);

		atomicAdd(&(s_dp[v1].x), -dp.x);
		atomicAdd(&(s_dp[v1].y), -dp.y);
		atomicAdd(&(s_dp[v1].z), -dp.z);
	}
	__syncthreads();

	// Calculate and store new mass states
	float3 newPos3;
	for(i = tid; i < cSimOpt.massesPerBlock && (i+massOffset) < cSimOpt.maxMasses; i+=stride) {
		newPos3 = s_pos[i];
		newPos3.x += s_dp[i].x;
		newPos3.y += s_dp[i].y;
		newPos3.z += s_dp[i].z;
		pos[i] = {newPos3.x, newPos3.y, newPos3.z, 0.0f};
		// vel[i].x = 0.99*(newPos3.x - pos4.x) / cSimOpt.dt;
		// vel[i].y = 0.99*(newPos3.y - pos4.y) / cSimOpt.dt;
		// vel[i].z = 0.99*(newPos3.z - pos4.z) / cSimOpt.dt;
	}
}

// __global__
// inline void update(float4 *__restrict__ pos, float4 *__restrict__ newPos, float4 *__restrict__ vel) {
// 	// Calculate and store new mass states
// 	int stride = blockDim.x * gridDim.x;

// 	float4 newPos4, pos4;
// 	for(uint i = blockIdx.x * blockDim.x + threadIdx.x; i < cSimOpt.maxMasses; i+=stride) {
// 		pos4 = __ldg(&pos[i]);
// 		newPos4 = __ldg(&newPos[i]);
// 		pos[i] = newPos4;
// 		vel[i].x = 0.99*(newPos4.x - pos4.x) / cSimOpt.dt;
// 		vel[i].y = 0.99*(newPos4.y - pos4.y) / cSimOpt.dt;
// 		vel[i].z = 0.99*(newPos4.z - pos4.z) / cSimOpt.dt;
// 	}
// }

void integrateBodies(DeviceData deviceData, uint numElements,
	SimOptions opt, 
	float time, uint step, bool integrateForce
	) {
	// Calculate and store new mass states
	
	uint numThreadsPerBlockDrag = 1024;
	uint numThreadsPerBlockPreSolve = 256;
	uint numThreadsPerBlockSolve = 1024;
	uint numThreadsPerBlockUpdate = 256;
	
	/*
	Notes on SM resources:
	thread blocks	: 8
	threads			: 2048
	registers		: 65536
	shared mem		: 49152
	*/

	uint maxSharedMemSize = 49152;
	uint bytesPerMass = sizeof(float3) + sizeof(float3);
	uint sharedMemSizeSolve = opt.massesPerBlock * bytesPerMass;
	uint numBlocksSolve = numElements;

	assert(sharedMemSizeSolve <= maxSharedMemSize);

	uint bytesPerBoundaryMass = sizeof(float3) + sizeof(float3) + sizeof(float3);
	uint sharedMemSizeDrag = opt.boundaryMassesPerBlock * bytesPerBoundaryMass;
	uint numBlocksDrag = numElements;

	assert(sharedMemSizeDrag <= maxSharedMemSize);

	uint numBlocksPreSolve = (opt.maxMasses + numThreadsPerBlockPreSolve - 1) / numThreadsPerBlockPreSolve;
	uint numBlocksUpdate = (opt.maxMasses + numThreadsPerBlockUpdate - 1) / numThreadsPerBlockUpdate;

	if(opt.boundaryMassesPerBlock > 0) {
		surfaceDragForce<<<numBlocksDrag,numThreadsPerBlockDrag,sharedMemSizeDrag>>>(
			(float4*) deviceData.dPos, (float4*) deviceData.dNewPos, 
			(float4*) deviceData.dVel, (ushort4*) deviceData.dFaces);
	}
	// pointDragForce<<<numBlocksPreSolve,numThreadsPerBlockPreSolve>>>(
	// 	(float4*) deviceData.dPos, (float4*) deviceData.dNewPos, 
	// 	(float4*) deviceData.dVel);
	hipDeviceSynchronize();

	// preSolve<<<numBlocksPreSolve, numThreadsPerBlockPreSolve>>>(
	// 	(float4*) deviceData.dPos, (float4*) deviceData.dNewPos,
	// 	(float4*) deviceData.dVel);
	// hipDeviceSynchronize();

	solveDistance<<<numBlocksSolve,numThreadsPerBlockSolve,sharedMemSizeSolve>>>(
		(float4*) deviceData.dPos,
		(float4*) deviceData.dNewPos,
		(float4*) deviceData.dVel,
		(ushort2*)  deviceData.dPairs, 
		(float*) deviceData.dSpringStresses,
		(uint8_t*) deviceData.dSpringMatIds,
		(float*) deviceData.dLbars,
		time, step, integrateForce);
	hipDeviceSynchronize();
		
	// update<<<numBlocksUpdate,numThreadsPerBlockUpdate>>>(
	// 	(float4*) deviceData.dPos,
	// 	(float4*) deviceData.dNewPos,
	// 	(float4*) deviceData.dVel);
	// hipDeviceSynchronize();
}

}