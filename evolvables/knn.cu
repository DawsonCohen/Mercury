#include "hip/hip_runtime.h"
#include "knn.h"
#include "knn_kernel.cu"
#include "mass.h"

#include <iostream>
#include <sys/stat.h>
#include <chrono>

#include <cub/device/device_segmented_radix_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>

#define BLOCK_SIZE 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

namespace KNN {

template<typename T>
std::vector<std::vector<std::vector<std::pair<unsigned int,float>>>> Batch(const std::vector<T>& mass_groups, unsigned int K)
{
    unsigned int num_groups = mass_groups.size();
    unsigned int masses_per_group = mass_groups[0].masses.size();
    unsigned int num_masses = num_groups * masses_per_group;

    // CPU data
    float* h_points = new float[num_masses * 3];
    unsigned int* h_indices = new unsigned int[num_masses * K];
    float* h_distances = new float[num_masses * K];

    for(unsigned int i = 0; i < num_groups; i++) {
        std::vector<Mass> masses = mass_groups[i].masses;
        for (unsigned int i = 0; i < masses_per_group; i++)
        {
            h_points[3*i]   = masses[i].pos[0];
            h_points[3*i+1] = masses[i].pos[1];
            h_points[3*i+2] = masses[i].pos[2];
        }
    }

    // GPU data 
    float* d_points;
    hipMalloc(&d_points, num_masses * 3 * sizeof(float));

    unsigned int* d_indices;
    hipMalloc(&d_indices, num_masses * K * sizeof(unsigned int));

    float* d_distances;
    hipMalloc(&d_distances, num_masses * K * sizeof(float));

    // Copy data to device memory
    hipMemcpy(d_points, h_points, num_masses * 3 * sizeof(float), hipMemcpyHostToDevice);

    // Shared memory size
    // unsigned int bytesPerThread = (sizeof(float) + sizeof(unsigned int)) * K;
    // unsigned int sharedMemSize = BLOCK_SIZE * bytesPerThread;
    
    // Compute k-nearest neighbors
    int num_blocks = num_groups;

    // printf("Num blocks: %u\n", num_blocks);
    // printf("Block size: %u\n", BLOCK_SIZE);
    // printf("Shared memory size: %u\n", sharedMemSize);
    
    // Execute the kernel
    k_nearest_neighbors_kernel<<<num_blocks, BLOCK_SIZE>>>((float3*) d_points, d_indices, d_distances, masses_per_group, K);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();


    hipMemcpy(h_indices, d_indices, num_masses * K * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(h_distances, d_distances, num_masses * K * sizeof(float), hipMemcpyDeviceToHost);


    std::vector<std::vector<std::vector<std::pair<unsigned int,float>>>> KNN(
        num_groups, std::vector<std::vector<std::pair<unsigned int, float>>>(masses_per_group, std::vector<std::pair<unsigned int,float>>(K)));
        
    for(unsigned int i = 0; i < num_groups; i++) {
        for(unsigned int j = 0; j < masses_per_group; j++) {
            for(unsigned int k = 0; k < K; k++) {
                int index = i*masses_per_group + j * K + k;
                KNN[i][j][k] = {h_indices[index], h_distances[index]};
            }
        }
    }

    // Free memory
    delete[] h_points;
    delete[] h_indices;
    delete[] h_distances;

    hipFree(d_points);
    hipFree(d_indices);
    hipFree(d_distances);
    
    return KNN;
}

void key_value_sort(uint* d_keys_in, uint* d_keys_out, float* d_values_in, float* d_values_out, uint count) {
    // Determine number of segments
    int num_segments = count;

    // Allocate memory on device for offsets
    int* h_offsets = new int[num_segments+1];
    for(uint i = 0; i < num_segments+1; i++) {
        h_offsets[i] = count*i;
    }

    int* d_offsets;
    hipMalloc(&d_offsets, (num_segments+1) * sizeof(int));
    hipMemcpy(d_offsets, h_offsets, (num_segments+1) *sizeof(int), hipMemcpyHostToDevice);

    // Determine temporary storage size
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedRadixSort::SortPairs(
        d_temp_storage, temp_storage_bytes,
        d_values_in, d_values_out, d_keys_in, d_keys_out,
        count*count, num_segments, d_offsets, d_offsets+1);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Run sorting operation
    hipcub::DeviceSegmentedRadixSort::SortPairs(
        d_temp_storage, temp_storage_bytes,
        d_values_in, d_values_out, d_keys_in, d_keys_out,
        count*count, count, d_offsets, d_offsets+1);

    delete[] h_offsets;
    hipFree(d_offsets);
}

template<typename T>
std::vector<std::vector<std::pair<unsigned int,float>>> KNN(const T& mass_group, unsigned int K)
{
    unsigned int num_masses = mass_group.masses.size();
    
    // CPU data
    float* h_points = new float[num_masses * 3];
    uint* h_ids = new uint[num_masses * num_masses];
    float* h_distances = new float[num_masses * num_masses];

    std::vector<Mass> masses = mass_group.masses;
    for (unsigned int i = 0; i < num_masses; i++)
    {
        h_points[3*i]   = masses[i].pos[0];
        h_points[3*i+1] = masses[i].pos[1];
        h_points[3*i+2] = masses[i].pos[2];
    }

    // GPU data 
    float* d_points;
    hipMalloc(&d_points, num_masses * 3 * sizeof(float));

    uint* d_ids;
    uint* d_ids_sorted;
    hipMalloc(&d_ids, num_masses * num_masses * sizeof(uint));
    hipMalloc(&d_ids_sorted, num_masses * num_masses * sizeof(uint));

    float* d_distances;
    float* d_distances_sorted;
    hipMalloc(&d_distances, num_masses * num_masses * sizeof(float));
    hipMalloc(&d_distances_sorted, num_masses * num_masses * sizeof(float));

    // Copy data to device memory
    hipMemcpy(d_points, h_points, num_masses * 3 * sizeof(float), hipMemcpyHostToDevice);

    // Compute k-nearest neighbors
    uint block_count = (num_masses + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 num_blocks = {block_count, block_count};

    // Execute the kernel
    distance_matrix_kernel<<<num_blocks, {BLOCK_SIZE, BLOCK_SIZE}>>>((float3*) d_points, d_ids, d_distances, num_masses);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    key_value_sort(d_ids, d_ids_sorted, d_distances, d_distances_sorted, num_masses);

    hipMemcpy(h_distances, d_distances_sorted, num_masses * num_masses * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_ids, d_ids_sorted, num_masses * num_masses * sizeof(uint), hipMemcpyDeviceToHost);


    std::vector<std::vector<float>> distances(num_masses, std::vector<float>(num_masses));
    for(uint i = 0; i < num_masses; i++) {
        for(uint j = 0; j < num_masses; j++) {
            distances[i][j] = h_distances[num_masses*i + j];
        }
    }
    
    
    std::vector<std::vector<std::pair<unsigned int,float>>> KNN(
        num_masses, std::vector<std::pair<unsigned int, float>>(K));

    for(uint i = 0; i < num_masses; i++) {
        for (uint j = 0; j < K; j++) {
            KNN[i][j].first = h_ids[i*num_masses + j];
            KNN[i][j].second = h_distances[i*num_masses + j];
        }
    }
    
    // Free memory
    delete[] h_points;
    delete[] h_ids;
    delete[] h_distances;

    hipFree(d_points);
    hipFree(d_ids);
    hipFree(d_ids_sorted);
    hipFree(d_distances);
    hipFree(d_distances_sorted);

    return KNN;
}


template<typename T>
std::vector<std::vector<std::pair<unsigned int,float>>> KNN_CPU(const T& mass_group, unsigned int K)
{
    unsigned int num_masses = mass_group.masses.size();

    std::vector<Mass> masses = mass_group.masses;

    std::vector<std::vector<float>> distances(num_masses, std::vector<float>(num_masses));
    for(uint i = 0; i < num_masses; i++) {
        Eigen::Vector3f p1 = masses[i].pos;
        for(uint j = 0; j < num_masses; j++) {
            Eigen::Vector3f p2 = masses[j].pos;
            distances[i][j] = distances[j][i] = (p1-p2).norm();
        }
    }

    std::vector<std::vector<std::pair<unsigned int,float>>> KNN(
        num_masses, std::vector<std::pair<unsigned int, float>>(K));

    for(uint i = 0; i < distances.size(); i++) {
        std::vector<std::pair<uint, float>> neighbors(distances.size());
        for (uint j = 0; j < distances.size(); j++) {
            if(distances[i][j] == 0.0f)
                neighbors[j] = {j, std::numeric_limits<double>::infinity()};
            else
                neighbors[j] = {j, distances[i][j]};
        }
        sort(neighbors.begin(), neighbors.end(), [](const std::pair<uint, float>& a, const std::pair<uint, float>& b) {
            return a.second < b.second;
        });
        neighbors.resize(K);

        KNN[i] = neighbors;
    }

    return KNN;
}

// Explicit instantiation of bar for NNRobot
template std::vector<std::vector<std::vector<std::pair<unsigned int,float>>>> Batch<NNRobot>(const std::vector<NNRobot>& mass_groups, unsigned int K);
template std::vector<std::vector<std::pair<unsigned int,float>>> KNN<NNRobot>(const NNRobot& mass_group, unsigned int K);
template std::vector<std::vector<std::pair<unsigned int,float>>> KNN_CPU<NNRobot>(const NNRobot& mass_group, unsigned int K);

}