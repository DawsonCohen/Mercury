#include "hip/hip_runtime.h"
#ifndef __SIM_KERNEL_H__
#define __SIM_KERNEL_H__

#include "vec_math.cuh"
#include <math.h>

// Explicity assumes each mass is of unit 1 mass
__device__
inline float3 gravityForce(float g) {
	return {0, -g, 0};
}

/*__device__
inline float3 collisionForce(float3 pos, float4 vel, float3 force,
					float4 env) {
	if(pos.y > 0.0f) return force;
	
	float3 Fc = {0.0f, 0.0f, 0.0f}; // force due to collision
	float3 Ff = {0.0f, 0.0f, 0.0f}; // force due to friction
	float magFc, magFf;

	Fc.y = env.x * (0.0f - pos.y);

	magFc = l2norm(force);
	magFf = env.y * Fc.y;

	//Static Friction
	if(vel.x == 0.0f && vel.z == 0.0f) {
		//Check if object has acceleration
		// TODO: check within epsilon
		if(magFc != 0) {
			if(magFc < magFf) {
				Ff = {-force.x, 0.0f, -force.y};
			} else {
				//Calculate direction of force and apply friction opposite based on magnitude
				Ff = magFf*normalize(force);
			}
		}
	} else {
		// Kinetic Friction
		Ff = magFf * normalize(make_float3(-vel.x, 0.0f, -vel.z));
	}

	force.x = Fc.x + Ff.x;
	force.y = Fc.y;
	force.z = Fc.z + Ff.z;

	return force;
}*/

__device__
float3 dragForce(float4 vel, float3 force,
					float rho) {	
	float3 Fd = {0.0f, 0.0f, 0.0f}; //Force due to drag - 1/2 * rho * v^2 * A * Cd (Assume A and Cd are 1)
	Fd.x = 1/2*vel.x*vel.x*rho;
	Fd.y = 1/2*vel.y*vel.y*rho;
	Fd.z = 1/2*vel.z*vel.z*rho;
	force.x += Fd.x;
	force.y += Fd.y;
	force.z += Fd.z;
	return force;
}

/*
	env: float4 describing global environment variables
		x - k		stiffness
		//y - mu		coefficient of friction
		y - rho		coefficient of drag
		z - zeta	damping
		w - g		acceleration due to gravity
*/
__device__
inline float3 environmentForce(float3 pos, float4 vel, float3 force,
						float4 env) {
	force = gravityForce(env.w);
	//force = collisionForce(pos,vel,force,env);
	force = dragForce(vel,force,env.y)
	return force;
}

/*
	mat: float4 describing spring material
		x - k		stiffness
		y - dL0 	maximum percent change
		z - omega	frequency of oscillation
		w - phi		phase
*/
__device__
inline float3 springForce(float3 bl, float3 br, float4 mat, 
					float mean_length, float time,
					float3 &force, float &magF)
{
	if(mat.x == 0.0f) {
		force = {0.0f, 0.0f, 0.0f};
		magF = 0.0f;
		return force;
	}

	float3	dir, diff;

	float	relative_change,
			rest_length,
			L;

	// b0pos = {bl.x, bl.y, bl.z};
	// b1pos = {br.x, br.y, br.z};

	// rest_length = mean_length * (1 + relative_change);
	relative_change = mat.y * sinf(mat.z*time+mat.w);
	rest_length = __fmaf_rn(mean_length, relative_change, mean_length);
	
	// rest_length = __fmaf_rn(mean_length*mat.y, sinf(mat.z*time+mat.w), mean_length);
	
	diff.x = bl.x - br.x;
	diff.y = bl.y - br.y;
	diff.z = bl.z - br.z;

	L = l2norm(diff);
	dir = {
		__fdiv_rn(diff.x,L),
		__fdiv_rn(diff.y,L),
		__fdiv_rn(diff.z,L)
	};

	magF = mat.x*(rest_length-L);

	force = magF * dir;
	
	return force;
}

struct SimOptions {
	float dt;
	float4 env;
	uint massesPerBlock;
	uint springsPerBlock;
	uint maxMasses;
	uint maxSprings;
	short shiftskip;
};

// TODO: test with single spring
__global__ void
inline integrateBodies(float4 *__restrict__ newPos, float4 *__restrict__ newVel,
				float4 *__restrict__ oldPos, float4 *__restrict__ oldVel,
				ushort2 *__restrict__ pairs, float4 *__restrict__ mats, float *__restrict__ Lbars,
				ushort *__restrict__ maxStressCount, ushort *__restrict__ minStressCount,
				float *__restrict__ stresses, uint *__restrict__ ids,
				float time, uint step, SimOptions opt)
{
	extern __shared__ float3 s[];
	float3  *s_pos = s;
	float3  *s_force = (float3*) &s_pos[opt.massesPerBlock];

	#ifdef STRESS_COUNT
	__shared__ ushort maxStressedSprings[1024];
	__shared__ ushort minStressedSprings[1024];
	#endif
	
	uint massOffset   = blockIdx.x * opt.massesPerBlock;
	uint springOffset = blockIdx.x * opt.springsPerBlock;

	int tid    = threadIdx.x;
	int stride = blockDim.x;
	
	// Initialize and compute environment forces
	float4 pos4;
	for(uint i = tid; i < opt.massesPerBlock && (i+massOffset) < opt.maxMasses; i+=stride) {
		pos4 = oldPos[i+massOffset];
		s_pos[i] = {pos4.x,pos4.y,pos4.z};
	}
	
	for(uint i = tid; i < opt.massesPerBlock && (i+massOffset) < opt.maxMasses; i+=stride) {
		s_force[i] = environmentForce(s_pos[i],oldVel[i+massOffset],s_force[i],opt.env);
	}
	__syncthreads();

	float4	mat;
	float3	bl, br;
	float3	force;
	ushort2	pair;
	float	Lbar,
			magF;
	ushort	left, right;

	#ifdef STRESS_COUNT
	float	minStress = 0.0f,
			maxStress  = 0.0f,
			nextMinStress = 0.0f;
	ushort	maxSpringIdx = tid,
			minSpringIdx = tid,
			nextMinSpringIdx = tid+stride,
			nextGroup_MaxSpringIdx,
			nextGroup_MinSpringIdx;
	#endif
	
	uint i;
	for(i = tid; i < opt.springsPerBlock && (i+springOffset) < opt.maxSprings; i+=stride) {
		pair = __ldg(&pairs[i+springOffset]);
		left  = pair.x;
		right = pair.y;
		bl = s_pos[left];
		br = s_pos[right];
		mat = __ldg(&mats[i+springOffset]);
		Lbar = __ldg(&Lbars[i+springOffset]);
		springForce(bl,br,mat,Lbar,time, force, magF);

		// if(fabsf(magF) > 0.0f) {
		atomicAdd(&(s_force[left].x), force.x);
		atomicAdd(&(s_force[left].y), force.y);
		atomicAdd(&(s_force[left].z), force.z);

		atomicAdd(&(s_force[right].x), -force.x);
		atomicAdd(&(s_force[right].y), -force.y);
		atomicAdd(&(s_force[right].z), -force.z);
		// }

		#ifdef STRESS_COUNT
		if(step % (opt.shiftskip+1) == 0) {
			if(fabsf(magF) > maxStress) {
				maxStress = fabsf(magF);
				maxSpringIdx = i;
			}
			if(((fabsf(magF) < minStress) || (minStress == 0.0f)) && fabsf(magF) > 0.0f) {
				if(i > tid) {
					nextMinStress = minStress;
					nextMinSpringIdx = minSpringIdx;
				}

				minStress = fabsf(magF);
				minSpringIdx = i;
				
			}
		}
		#endif

		#ifdef FULL_STRESS
		stresses[i + springOffset] = stresses[i + springOffset] + fabsf(magF);
		#endif
	}

	#ifdef STRESS_COUNT
	ushort2 cMaxPair,
			cMinPair;
	ushort	cMax_MaxCount,
			cMax_MinCount,
			cMin_MaxCount,
			cMin_MinCount;
	float4	cMaxMat,
			cMinMat;
	float	cMaxLbar,
			cMinLbar;
	#ifdef FULL_STRESS
	float	cMaxStress,
			cMinStress;
	float	cMaxID,
			cMinID;
	#endif

	
	if(minSpringIdx == maxSpringIdx) { // guarentees not both max and min
		minSpringIdx = nextMinSpringIdx;
		minStress = nextMinStress;
	}

	if(step % (opt.shiftskip+1) == 0) {
		maxStressCount[maxSpringIdx + springOffset] += (maxStress > 0.0f);
		minStressCount[minSpringIdx + springOffset] += (minStress > 0.0f);
		maxStressedSprings[tid]  = maxSpringIdx;
		minStressedSprings[tid]  = minSpringIdx;
		
		// current thread max spring info
		cMaxPair = pairs[maxSpringIdx + springOffset];
		cMax_MaxCount = maxStressCount[maxSpringIdx + springOffset];
		cMax_MinCount = minStressCount[maxSpringIdx + springOffset];
		cMaxMat = mats[maxSpringIdx + springOffset];
		cMaxLbar = Lbars[maxSpringIdx + springOffset];

		cMinPair = pairs[minSpringIdx + springOffset];
		cMin_MaxCount = maxStressCount[minSpringIdx + springOffset];
		cMin_MinCount = minStressCount[minSpringIdx + springOffset];
		cMinMat = mats[minSpringIdx + springOffset];
		cMinLbar = Lbars[minSpringIdx + springOffset];

		#ifdef FULL_STRESS
		cMaxStress = stresses[maxSpringIdx + springOffset];
		cMaxID = ids[maxSpringIdx + springOffset];
		cMinStress = stresses[minSpringIdx + springOffset];
		cMinID = ids[minSpringIdx + springOffset];
		#endif
	}
	#endif
	__syncthreads();

	#ifdef STRESS_COUNT
	int tid_next = (tid+1) % blockDim.x;
	if(step % (opt.shiftskip+1) == 0) {
		// shift current index to next spring
		nextGroup_MaxSpringIdx  = maxStressedSprings[tid_next];
		nextGroup_MinSpringIdx = minStressedSprings[tid_next];

		pairs[nextGroup_MaxSpringIdx + springOffset] = cMaxPair;
		mats[nextGroup_MaxSpringIdx + springOffset] = cMaxMat;
		Lbars[nextGroup_MaxSpringIdx + springOffset] = cMaxLbar;
		maxStressCount[nextGroup_MaxSpringIdx + springOffset] = cMax_MaxCount;
		minStressCount[nextGroup_MaxSpringIdx + springOffset] = cMax_MinCount;

		pairs[nextGroup_MinSpringIdx + springOffset] = cMinPair;
		mats[nextGroup_MinSpringIdx + springOffset] = cMinMat;
		Lbars[nextGroup_MinSpringIdx + springOffset] = cMinLbar;
		maxStressCount[nextGroup_MinSpringIdx + springOffset] = cMin_MaxCount;
		minStressCount[nextGroup_MinSpringIdx + springOffset] = cMin_MinCount;

		#ifdef FULL_STRESS
		stresses[nextGroup_MaxSpringIdx + springOffset] = cMaxStress;
		ids[nextGroup_MaxSpringIdx + springOffset] = cMaxID;

		stresses[nextGroup_MinSpringIdx + springOffset] = cMinStress;
		ids[nextGroup_MinSpringIdx + springOffset] = cMinID;
		#endif
	}
	#endif

	// Calculate and store new mass states
	float4 vel;
	float3 pos3;
	for(uint i = tid; i < opt.massesPerBlock && (i+massOffset) < opt.maxMasses; i+=stride) {
		vel = oldVel[i+massOffset];

		vel.x += (s_force[i].x * opt.dt)*opt.env.z;
		vel.y += (s_force[i].y * opt.dt)*opt.env.z;
		vel.z += (s_force[i].z * opt.dt)*opt.env.z;

		// new position = old position + velocity * deltaTime
		s_pos[i].x += vel.x * opt.dt;
		s_pos[i].y += vel.y * opt.dt;
		s_pos[i].z += vel.z * opt.dt;

		// store new position and velocity
		pos3 = s_pos[i];
		newPos[i+massOffset] = {pos3.x, pos3.y, pos3.z};
		newVel[i+massOffset] = vel;
	}
}

#endif
