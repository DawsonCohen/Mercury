#include "hip/hip_runtime.h"
#ifndef __SIM_KERNEL_CUH__
#define __SIM_KERNEL_CUH__

#include "vec_math.cuh"
#include <math.h>
#include <assert.h>
#include <stdio.h>

#define EPS (float) 0.000001
#define MIN_DIST (float) 0.005
#define MAX_FORCE (float) 200000

// Explicity assumes each mass is of unit 1 mass
__device__
inline void gravityForce(float3& force) {
	force = {0, -9.81f, 0};
}

__device__
inline void collisionForce(float3 pos, float4 vel, float3& force,
					float4 env) {
	if(pos.y > 0.0f) return;
	
	float3 Fc = {0.0f, 0.0f, 0.0f}; // force due to collision
	float3 Ff = {0.0f, 0.0f, 0.0f}; // force due to friction
	float magFc, magFf;

	Fc.y = env.x * (0.0f - pos.y);

	magFc = l2norm(force);
	magFf = env.y * Fc.y;

	//Static Friction
	if(fabsf(vel.x) < EPS && fabsf(vel.z) < EPS) {
		//Check if object has acceleration
		if(fabsf(magFc) > EPS) {
			if(magFc < magFf) {
				Ff = {-force.x, 0.0f, -force.y};
			} else {
				//Calculate direction of force and apply friction opposite based on magnitude
				Ff = magFf*normalize(force);
			}
		}
	} else {
		// Kinetic Friction
		Ff = magFf * normalize(make_float3(-vel.x, 0.0f, -vel.z));
	}

	force.x = Fc.x + Ff.x;
	force.y = Fc.y;
	force.z = Fc.z + Ff.z;
}

/*
	env: float4 describing global environment variables
		x - k		stiffness
		y - mu		coefficient of friction
		z - zeta	damping
		w - g		acceleration due to gravity
*/
__device__
inline void environmentForce(float3 pos, float4 vel, float3& force,
						float4 env) {
	gravityForce(force);
	// assert(!isnan(force.x) && !isnan(force.y) && !isnan(force.z));
	collisionForce(pos,vel,force,env);
	// assert(!isnan(force.x) && !isnan(force.y) && !isnan(force.z));
}

/*
	mat: float4 describing spring material
		x - k		stiffness
		y - dL0 	maximum percent change
		z - omega	frequency of oscillation
		w - phi		phase
*/
__device__
inline void springForce(float3 bl, float3 br, float4 mat, 
					float mean_length, float time,
					float3& force, float& magF)
{
	if(mat.x == 0.0f) {
		force = {0.0f, 0.0f, 0.0f};
		magF = 0.0f;
		return;
	}

	float3	dir, diff;

	float	relative_change,
			rest_length,
			L;

	// b0pos = {bl.x, bl.y, bl.z};
	// b1pos = {br.x, br.y, br.z};

	// rest_length = mean_length * (1 + relative_change);
	relative_change = mat.y * sinf(mat.z*time+mat.w);
	rest_length = __fmaf_rn(mean_length, relative_change, mean_length);
	
	// rest_length = __fmaf_rn(mean_length*mat.y, sinf(mat.z*time+mat.w), mean_length);
	
	diff.x = bl.x - br.x;
	diff.y = bl.y - br.y;
	diff.z = bl.z - br.z;

	L = l2norm(diff);
	dir = {
		__fdiv_rn(diff.x,L),
		__fdiv_rn(diff.y,L),
		__fdiv_rn(diff.z,L)
	};
	if(isnan(dir.x) || isnan(dir.y) || isnan(dir.z)) {
		force = {0.0f, 0.0f, 0.0f};
	} else {
		magF = min(mat.x*(rest_length-L), MAX_FORCE);
		force = magF * dir;
	}

	// assert(!isnan(force.x) && !isnan(force.y) && !isnan(force.z));
}

struct SimOptions {
	float dt;
	float4 env;
	uint massesPerBlock;
	uint springsPerBlock;
	uint maxMasses;
	uint maxSprings;
	short shiftskip;
};

// TODO: test with single spring
__global__ void
inline integrateBodies(float4 *__restrict__ newPos, float4 *__restrict__ newVel,
				float4 *__restrict__ oldPos, float4 *__restrict__ oldVel,
				ushort2 *__restrict__ pairs, float4 *__restrict__ mats, float *__restrict__ Lbars,
				ushort *__restrict__ maxStressCount, ushort *__restrict__ minStressCount,
				float *__restrict__ stresses, uint *__restrict__ ids,
				float time, uint step, SimOptions opt)
{
	extern __shared__ float3 s[];
	float3  *s_pos = s;
	float3  *s_force = (float3*) &s_pos[opt.massesPerBlock];
	
	uint massOffset   = blockIdx.x * opt.massesPerBlock;
	uint springOffset = blockIdx.x * opt.springsPerBlock;

	int tid    = threadIdx.x;
	int stride = blockDim.x;
	
	// Initialize and compute environment forces
	float4 pos4;
	for(uint i = tid; i < opt.massesPerBlock && (i+massOffset) < opt.maxMasses; i+=stride) {
		pos4 = oldPos[i+massOffset];
		s_pos[i] = {pos4.x,pos4.y,pos4.z};
	}
	
	for(uint i = tid; i < opt.massesPerBlock && (i+massOffset) < opt.maxMasses; i+=stride) {
		environmentForce(s_pos[i],oldVel[i+massOffset],s_force[i],opt.env);
	}
	__syncthreads();

	float4	mat;
	float3	bl, br;
	float3	force;
	ushort2	pair;
	float	Lbar,
			magF;
	ushort	left, right;
	
	uint i;
	for(i = tid; i < opt.springsPerBlock && (i+springOffset) < opt.maxSprings; i+=stride) {
		pair = __ldg(&pairs[i+springOffset]);
		left  = pair.x;
		right = pair.y;
		bl = s_pos[left];
		br = s_pos[right];
		mat = __ldg(&mats[i+springOffset]);
		Lbar = __ldg(&Lbars[i+springOffset]);
		springForce(bl,br,mat,Lbar,time, force, magF);
		// assert(!isnan(force.x) && !isnan(force.y) && !isnan(force.z));

		atomicAdd(&(s_force[left].x), force.x);
		atomicAdd(&(s_force[left].y), force.y);
		atomicAdd(&(s_force[left].z), force.z);

		atomicAdd(&(s_force[right].x), -force.x);
		atomicAdd(&(s_force[right].y), -force.y);
		atomicAdd(&(s_force[right].z), -force.z);
	}
	__syncthreads();

	// Calculate and store new mass states
	float4 vel;
	float3 pos3;
	for(uint i = tid; i < opt.massesPerBlock && (i+massOffset) < opt.maxMasses; i+=stride) {
		vel = oldVel[i+massOffset];

		vel.x += (s_force[i].x * opt.dt)*opt.env.z;
		vel.y += (s_force[i].y * opt.dt)*opt.env.z;
		vel.z += (s_force[i].z * opt.dt)*opt.env.z;

		// assert(!isnan(vel.x) && !isnan(vel.y) && !isnan(vel.z));

		// new position = old position + velocity * deltaTime
		s_pos[i].x += vel.x * opt.dt;
		s_pos[i].y += vel.y * opt.dt;
		s_pos[i].z += vel.z * opt.dt;

		// store new position and velocity
		// assert(!isnan(pos3.x) && !isnan(pos3.y) && !isnan(pos3.z));
		pos3 = s_pos[i];
		newPos[i+massOffset] = {pos3.x, pos3.y, pos3.z};
		newVel[i+massOffset] = vel;
	}
}


__global__ void
inline integrateBodiesStresses(float4 *__restrict__ newPos, float4 *__restrict__ newVel,
				float4 *__restrict__ oldPos, float4 *__restrict__ oldVel,
				ushort2 *__restrict__ pairs, float4 *__restrict__ mats, float *__restrict__ Lbars,
				ushort *__restrict__ maxStressCount, ushort *__restrict__ minStressCount,
				float *__restrict__ stresses, uint *__restrict__ ids,
				float time, uint step, SimOptions opt)
{
	extern __shared__ float3 s[];
	float3  *s_pos = s;
	float3  *s_force = (float3*) &s_pos[opt.massesPerBlock];

	__shared__ ushort maxStressedSprings[1024];
	__shared__ ushort minStressedSprings[1024];
	
	uint massOffset   = blockIdx.x * opt.massesPerBlock;
	uint springOffset = blockIdx.x * opt.springsPerBlock;

	int tid    = threadIdx.x;
	int stride = blockDim.x;
	
	// Initialize and compute environment forces
	float4 pos4;
	for(uint i = tid; i < opt.massesPerBlock && (i+massOffset) < opt.maxMasses; i+=stride) {
		pos4 = oldPos[i+massOffset];
		s_pos[i] = {pos4.x,pos4.y,pos4.z};
	}
	
	for(uint i = tid; i < opt.massesPerBlock && (i+massOffset) < opt.maxMasses; i+=stride) {
		environmentForce(s_pos[i],oldVel[i+massOffset],s_force[i],opt.env);
	}
	__syncthreads();

	float4	mat;
	float3	bl, br;
	float3	force;
	ushort2	pair;
	float	Lbar,
			magF;
	ushort	left, right;

	float	minStress = 0.0f,
			maxStress  = 0.0f,
			nextMinStress = 0.0f;
	ushort	maxSpringIdx = tid,
			minSpringIdx = tid,
			nextMinSpringIdx = tid+stride,
			nextGroup_MaxSpringIdx,
			nextGroup_MinSpringIdx;
	
	uint i;
	for(i = tid; i < opt.springsPerBlock && (i+springOffset) < opt.maxSprings; i+=stride) {
		pair = __ldg(&pairs[i+springOffset]);
		left  = pair.x;
		right = pair.y;
		bl = s_pos[left];
		br = s_pos[right];
		mat = __ldg(&mats[i+springOffset]);
		Lbar = __ldg(&Lbars[i+springOffset]);
		springForce(bl,br,mat,Lbar,time, force, magF);

		// assert(!isnan(force.x) && !isnan(force.y) && !isnan(force.z));

		atomicAdd(&(s_force[left].x), force.x);
		atomicAdd(&(s_force[left].y), force.y);
		atomicAdd(&(s_force[left].z), force.z);

		atomicAdd(&(s_force[right].x), -force.x);
		atomicAdd(&(s_force[right].y), -force.y);
		atomicAdd(&(s_force[right].z), -force.z);

		if(step % (opt.shiftskip+1) == 0) {
			if(fabsf(magF) > maxStress) {
				maxStress = fabsf(magF);
				maxSpringIdx = i;
			}
			if(((fabsf(magF) < minStress) || (minStress == 0.0f)) && fabsf(magF) > 0.0f) {
				if(i > tid) {
					nextMinStress = minStress;
					nextMinSpringIdx = minSpringIdx;
				}

				minStress = fabsf(magF);
				minSpringIdx = i;
				
			}
		}

		#ifdef FULL_STRESS
		stresses[i + springOffset] = stresses[i + springOffset] + fabsf(magF);
		#endif
	}

	ushort2 cMaxPair,
			cMinPair;
	ushort	cMax_MaxCount,
			cMax_MinCount,
			cMin_MaxCount,
			cMin_MinCount;
	float4	cMaxMat,
			cMinMat;
	float	cMaxLbar,
			cMinLbar;
	#ifdef FULL_STRESS
	float	cMaxStress,
			cMinStress;
	float	cMaxID,
			cMinID;
	#endif

	
	if(minSpringIdx == maxSpringIdx) { // guarentees not both max and min
		minSpringIdx = nextMinSpringIdx;
		minStress = nextMinStress;
	}

	if(step % (opt.shiftskip+1) == 0) {
		maxStressCount[maxSpringIdx + springOffset] += (maxStress > 0.0f);
		minStressCount[minSpringIdx + springOffset] += (minStress > 0.0f);
		maxStressedSprings[tid]  = maxSpringIdx;
		minStressedSprings[tid]  = minSpringIdx;
		
		// current thread max spring info
		cMaxPair = pairs[maxSpringIdx + springOffset];
		cMax_MaxCount = maxStressCount[maxSpringIdx + springOffset];
		cMax_MinCount = minStressCount[maxSpringIdx + springOffset];
		cMaxMat = mats[maxSpringIdx + springOffset];
		cMaxLbar = Lbars[maxSpringIdx + springOffset];

		cMinPair = pairs[minSpringIdx + springOffset];
		cMin_MaxCount = maxStressCount[minSpringIdx + springOffset];
		cMin_MinCount = minStressCount[minSpringIdx + springOffset];
		cMinMat = mats[minSpringIdx + springOffset];
		cMinLbar = Lbars[minSpringIdx + springOffset];

		#ifdef FULL_STRESS
		cMaxStress = stresses[maxSpringIdx + springOffset];
		cMaxID = ids[maxSpringIdx + springOffset];
		cMinStress = stresses[minSpringIdx + springOffset];
		cMinID = ids[minSpringIdx + springOffset];
		#endif
	}
	__syncthreads();

	int tid_next = (tid+1) % blockDim.x;
	if(step % (opt.shiftskip+1) == 0) {
		// shift current index to next spring
		nextGroup_MaxSpringIdx  = maxStressedSprings[tid_next];
		nextGroup_MinSpringIdx = minStressedSprings[tid_next];

		pairs[nextGroup_MaxSpringIdx + springOffset] = cMaxPair;
		mats[nextGroup_MaxSpringIdx + springOffset] = cMaxMat;
		Lbars[nextGroup_MaxSpringIdx + springOffset] = cMaxLbar;
		maxStressCount[nextGroup_MaxSpringIdx + springOffset] = cMax_MaxCount;
		minStressCount[nextGroup_MaxSpringIdx + springOffset] = cMax_MinCount;

		pairs[nextGroup_MinSpringIdx + springOffset] = cMinPair;
		mats[nextGroup_MinSpringIdx + springOffset] = cMinMat;
		Lbars[nextGroup_MinSpringIdx + springOffset] = cMinLbar;
		maxStressCount[nextGroup_MinSpringIdx + springOffset] = cMin_MaxCount;
		minStressCount[nextGroup_MinSpringIdx + springOffset] = cMin_MinCount;

		#ifdef FULL_STRESS
		stresses[nextGroup_MaxSpringIdx + springOffset] = cMaxStress;
		ids[nextGroup_MaxSpringIdx + springOffset] = cMaxID;

		stresses[nextGroup_MinSpringIdx + springOffset] = cMinStress;
		ids[nextGroup_MinSpringIdx + springOffset] = cMinID;
		#endif
	}

	// Calculate and store new mass states
	float4 vel;
	float3 pos3;
	for(uint i = tid; i < opt.massesPerBlock && (i+massOffset) < opt.maxMasses; i+=stride) {
		vel = oldVel[i+massOffset];

		vel.x += (s_force[i].x * opt.dt)*opt.env.z;
		vel.y += (s_force[i].y * opt.dt)*opt.env.z;
		vel.z += (s_force[i].z * opt.dt)*opt.env.z;

		// new position = old position + velocity * deltaTime
		s_pos[i].x += vel.x * opt.dt;
		s_pos[i].y += vel.y * opt.dt;
		s_pos[i].z += vel.z * opt.dt;

		// store new position and velocity
		pos3 = s_pos[i];
		newPos[i+massOffset] = {pos3.x, pos3.y, pos3.z};
		newVel[i+massOffset] = vel;
	}
}

#endif
