#include "hip/hip_runtime.h"
#include "Simulator.h"
#include "sim_kernel.cu"
#include <math.h>
#include <algorithm>
#include <functional>
#include <fstream>
#include <iostream>
#include <random>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

Simulator::Simulator(Element prototype, uint maxElements) :
	massesPerElement(prototype.masses.size()),
	springsPerElement(prototype.springs.size()),
	maxElements(maxElements),
	maxMasses(prototype.masses.size()*maxElements),
	maxSprings(prototype.springs.size()*maxElements),
	maxEnvs(1),
	m_hPos(0),
	m_hVel(0)
{
	m_dPos[0] = m_dPos[1] = 0;
    m_dVel[0] = m_dVel[1] = 0;
	
	_initialize();
}

Simulator::~Simulator() {
	// Free CPU
	delete[] m_hPos;
	delete[] m_hVel;

	delete[] m_hLbars;
	delete[] m_hPairs;
	delete[] m_hMats;

	delete[] massBuf;
	delete[] springBuf;
	delete[] envBuf;
	delete[] offsetBuf;

	delete[] m_hMaxStressCount;
	delete[] m_hMinStressCount;

	// Free GPU
	hipFree((void**) m_dPos[0]);
	hipFree((void**) m_dPos[1]);
	hipFree((void**) m_dVel[0]);
	hipFree((void**) m_dVel[1]);

	hipFree((void**) m_dLbars);
	hipFree((void**) m_dPairs);
	hipFree((void**) m_dMats);

	hipFree((void**) m_dMaxStressCount);
	hipFree((void**) m_dMinStressCount);
	hipFree((void**) m_dStresses);
	hipFree((void**) m_dSpringIDs);

	hipFree((void**) m_dMatEncodings);
}

void Simulator::Initialize(Element prototype, uint maxElements) {
	
	massesPerElement = prototype.masses.size();
	springsPerElement = prototype.springs.size();
	this->maxElements = maxElements;
	maxMasses = prototype.masses.size()*maxElements;
	maxSprings = prototype.springs.size()*maxElements;
	maxEnvs = 1;
	m_currentRead = 0;
	m_currentWrite = 1;

	_initialize();
}

void Simulator::_initialize() { //uint maxMasses, uint maxSprings) {
	if(initialized) {
		// Free CPU
		delete[] m_hPos;
		delete[] m_hVel;

		delete[] m_hLbars;
		delete[] m_hPairs;
		delete[] m_hMats;

		delete[] massBuf;
		delete[] springBuf;
		delete[] envBuf;
		delete[] offsetBuf;

		delete[] m_hMaxStressCount;
		delete[] m_hMinStressCount;

		delete[] m_hMatEncodings;

		// Free GPU
		hipFree((void**) m_dPos[0]);
		hipFree((void**) m_dPos[1]);
		hipFree((void**) m_dVel[0]);
		hipFree((void**) m_dVel[1]);

		hipFree((void**) m_dLbars);
		hipFree((void**) m_dPairs);
		hipFree((void**) m_dMats);

		hipFree((void**) m_dMaxStressCount);
		hipFree((void**) m_dMinStressCount);
		hipFree((void**) m_dStresses);
		hipFree((void**) m_dSpringIDs);

		hipFree((void**) m_dMatEncodings);
	}
	initialized = true;
	
	printf("Num Masses:\t%u\n",maxMasses);
	printf("Num Springs:\t%u\n",maxSprings);
	
	massBuf   = new Mass[maxMasses];
	springBuf = new Spring[maxSprings];
	offsetBuf = new uint[maxSprings];
	envBuf 	  =	new Environment[1];

	m_hLbars  = new float[maxSprings];
	m_hPairs  = new ushort[maxSprings*2];
	m_hMats   = new float[maxSprings*4];

	m_hPos 	  = new float[maxMasses*4];
    m_hVel 	  = new float[maxMasses*4];

	m_hMaxStressCount = new ushort[maxSprings];
	m_hMinStressCount  = new ushort[maxSprings];

	m_hStresses  = new float[maxSprings];
	m_hSpringIDs = new uint[maxSprings];

	m_hMatEncodings = new char[maxSprings];
	
	memset(m_hPos, 0, maxMasses*4*sizeof(float));
    memset(m_hVel, 0, maxMasses*4*sizeof(float));

	memset(m_hMaxStressCount, 0, maxSprings*sizeof(ushort));
    memset(m_hMinStressCount, 0, maxSprings*sizeof(ushort));

    memset(m_hStresses, 0, maxSprings * sizeof(float));
    memset(m_hSpringIDs, 0, maxSprings * sizeof(uint));

	memset(m_hMatEncodings, 0, maxSprings * sizeof(char));
	
    unsigned int massSizefloat4     = sizeof(float)  * 4 * maxMasses;
    unsigned int springSizeushort	= sizeof(ushort) * 1 * maxSprings;
    unsigned int springSizefloat    = sizeof(float)  * 1 * maxSprings;
    unsigned int springSizefloat4   = sizeof(float)  * 4 * maxSprings;
    unsigned int springSizeushort2  = sizeof(ushort) * 2 * maxSprings;
    unsigned int springSizeuint     = sizeof(uint)   * 1 * maxSprings;
    unsigned int springSizechar     = sizeof(char)   * 1 * maxSprings;
	
	hipMalloc((void**)&m_dVel[0], massSizefloat4);
	hipMalloc((void**)&m_dVel[1], massSizefloat4);

	hipMalloc((void**)&m_dPos[0], massSizefloat4);
	hipMalloc((void**)&m_dPos[1], massSizefloat4);

	hipMalloc((void**)&m_dPairs,  springSizeushort2);
	hipMalloc((void**)&m_dLbars,  springSizefloat);
	hipMalloc((void**)&m_dMats,   springSizefloat4);
	hipMalloc((void**)&m_dMats,   springSizefloat4);
	
	hipMalloc((void**)&m_dMaxStressCount, springSizeushort);
	hipMalloc((void**)&m_dMinStressCount,  springSizeushort);

	hipMalloc((void**)&m_dStresses,  springSizefloat);
	hipMalloc((void**)&m_dSpringIDs,  springSizeuint);

	hipMalloc((void**)&m_dMatEncodings,  springSizechar);

	envBuf[0] = Environment();
	envCount++;
}

std::vector<ElementTracker> Simulator::Simulate(std::vector<Element>& elements) {
	numMasses = 0; numSprings = 0; numElements = 0;

	std::vector<ElementTracker> trackers = Allocate(elements);
	
	float3 gravity = {envBuf[0].g.x(), envBuf[0].g.y(), envBuf[0].g.z()};
	float stiffness = envBuf[0].floor_stiffness;
	float mu = envBuf[0].friction;
	float zeta = envBuf[0].damping;
	float step_time = 0;
	Eigen::Vector3f pos, vel;
	for(uint i = 0; i < numMasses; i++) {
		float  mass = massBuf[i].mass;
		vel  = massBuf[i].vel;
		pos = massBuf[i].pos;

		m_hPos[4*i]   = pos.x();
		m_hPos[4*i+1] = pos.y();
		m_hPos[4*i+2] = pos.z();
		m_hPos[4*i+3] = mass;
		
		m_hVel[4*i]   = vel.x();
		m_hVel[4*i+1] = vel.y();
		m_hVel[4*i+2] = vel.z();
	}

	for(uint i = 0; i < numSprings; i++) {
		Material mat    = springBuf[i].material;
		float    lbar   = springBuf[i].mean_length;
		uint	 left   = springBuf[i].m0,
			     right  = springBuf[i].m1;

		// printf("%u:\t%u\n",offsetBuf[i],springBuf[i].m0);

		m_hMats[4*i]   = mat.k;
		m_hMats[4*i+1] = mat.dL0;
		m_hMats[4*i+2] = mat.omega;
		m_hMats[4*i+3] = mat.phi;

		m_hMatEncodings[i] = mat.encoding;

		m_hPairs[2*i]   = left;
		m_hPairs[2*i+1] = right;
		m_hLbars[i] 	= lbar;

		m_hSpringIDs[i] = i;
	}

	for(uint i = 0; i < ACTIVE_MATERIAL_COUNT; i++) {
		Material m = materials::matLookup(i);
		m_hMaterials[4*i]   = m.k;
		m_hMaterials[4*i+1] = m.dL0;
		m_hMaterials[4*i+2] = m.omega;
		m_hMaterials[4*i+3] = m.phi;
	}

	hipMemcpy(m_dVel[m_currentRead], m_hVel,   numMasses   *4*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(m_dPos[m_currentRead], m_hPos,   numMasses   *4*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(m_dPairs,  m_hPairs,  numSprings *2*sizeof(ushort),  hipMemcpyHostToDevice);
	hipMemcpy(m_dLbars,  m_hLbars,  numSprings  * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(m_dMats,   m_hMats,   numSprings *4*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(m_dMatEncodings,   m_hMatEncodings,   numSprings *sizeof(char), hipMemcpyHostToDevice);
	#ifdef FULL_STRESS
	hipMemcpy(m_dStresses,   m_hStresses,   numSprings*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(m_dSpringIDs,   m_hSpringIDs,   numSprings*sizeof(uint), hipMemcpyHostToDevice);
	#endif

	hipMemcpy(m_dMaxStressCount,   m_hMaxStressCount,   numSprings*sizeof(ushort), hipMemcpyHostToDevice);
	hipMemcpy(m_dMinStressCount,   m_hMinStressCount,   numSprings*sizeof(ushort), hipMemcpyHostToDevice);
	
	hipMemcpy(m_dMatEncodings, m_hMatEncodings, numSprings*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_materials), m_hMaterials, 4*4*sizeof(float));

	gpuErrchk( hipPeekAtLastError() );
	
	/*
	Notes on SM resources:
	thread blocks	: 8
	threads			: 2048
	registers		: 65536
	shared mem		: 49152
	*/

	uint maxSharedMemSize = 49152;
	uint bytesPerMass = sizeof(float3) + sizeof(float3);
	uint bytesPerElement = massesPerElement*bytesPerMass;
	// Must equal 1 for proper max/min spring calculation
	uint elementsPerBlock = min(maxSharedMemSize / bytesPerElement, numElements);
	uint massesPerBlock = massesPerElement * elementsPerBlock;
	uint springsPerBlock = springsPerElement * elementsPerBlock;
	uint sharedMemSize = massesPerBlock * bytesPerMass;
	int numBlocks = (numElements + elementsPerBlock - 1) / elementsPerBlock;

	assert(sharedMemSize < maxSharedMemSize);
	// uint bytesPerBlock = elementsPerBlock * bytesPerElement;
	// int numBlocks = (springsPerBlock + threadsPerBlock - 1) / threadsPerBlock;
	// printf("BPE:\t%u\n", bytesPerElement);
	// printf("Block Utilization:\t%f\n", (float) bytesPerBlock / (float) maxSharedMemSize);
	// printf("EPB:\t%u\n", elementsPerBlock);
	// printf("EPB:\t%u\n", elementsPerBlock);

	short shiftskip = 20;

	SimOptions opt = {
		step_period,
		make_float4(stiffness,mu,zeta,gravity.y),
		massesPerBlock, springsPerBlock,
		numMasses, numSprings,
		shiftskip
	};
	
	uint step = 0;
	float hold_time = 0.0f;
	float mat_time = 0.0f;
	while(step_time < max_time) {
		mat_time = max(total_time-hold_time,0.0f);

		integrateBodies<<<numBlocks,threadsPerBlock,sharedMemSize>>>(
			(float4*) m_dPos[m_currentWrite], (float4*) m_dVel[m_currentWrite],
			(float4*) m_dPos[m_currentRead], (float4*) m_dVel[m_currentRead],
			(ushort2*)  m_dPairs,
			(float*) m_dLbars,
			(float4*) m_dMats,
			// (char*) m_dMatEncodings,
			(ushort*) m_dMaxStressCount, (ushort*) m_dMinStressCount,
			#ifdef FULL_STRESS
			(float*) m_dStresses, 
			(uint*) m_dSpringIDs,
			#endif
			mat_time, step, opt);

			
		gpuErrchk( hipPeekAtLastError() );
		hipDeviceSynchronize();
		
		std::swap(m_currentRead, m_currentWrite);
		
		step++;
		total_time += step_period;
		step_time += step_period;
	}

	hipMemcpy(m_hPos,m_dPos[m_currentRead],numMasses*4*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(m_hVel,m_dVel[m_currentRead],numMasses*4*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(m_hMaxStressCount,m_dMaxStressCount,numSprings*sizeof(ushort),hipMemcpyDeviceToHost);
	hipMemcpy(m_hMinStressCount, m_dMinStressCount, numSprings*sizeof(ushort),hipMemcpyDeviceToHost);
	#ifdef FULL_STRESS
	hipMemcpy(m_hStresses,   m_dStresses,   numSprings*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(m_hSpringIDs,   m_dSpringIDs,   numSprings*sizeof(uint), hipMemcpyDeviceToHost);
	#endif

	for(uint i = 0; i < numMasses; i++) {
		float3 pos = {m_hPos[4*i], m_hPos[4*i+1], m_hPos[4*i+2]};
		float3 vel = {m_hVel[4*i], m_hVel[4*i+1], m_hVel[4*i+2]};
		massBuf[i].pos = Eigen::Vector3f(pos.x,pos.y,pos.z);

		// printf("%u: {%f,%f,%f}\n",i,pos.x,pos.y,pos.z);
		massBuf[i].vel = Eigen::Vector3f(vel.x,vel.y,vel.z);
	}

	#if defined(FULL_STRESS) && defined(WRITE_STRESS)
	std::vector<std::tuple<uint, float, uint, uint>> stressHistory;

	for(uint i = 0; i < maxSprings; i++) {
		stressHistory.push_back({m_hSpringIDs[i], m_hStresses[i], m_hMaxStressCount[i], m_hMinStressCount[i]});
	}
	// std::string stressHistoryCSV = util::DataToCSV("id, stress, max count, min count",stressHistory);
	// util::WriteCSV("../z_results/stress.csv", stressHistoryCSV);
	#endif

	hipMemcpy(m_dVel[m_currentRead], m_hVel,   numMasses   *4*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(m_dPos[m_currentRead], m_hPos,   numMasses   *4*sizeof(float), hipMemcpyHostToDevice);

	return trackers;
}


std::vector<ElementTracker> Simulator::Allocate(const std::vector<Element>& elements) {
	std::vector<ElementTracker> trackers;

	for(uint i = 0; i < elements.size(); i++) {
		trackers.push_back(AllocateElement(elements[i]));
	}

	return trackers;
}

ElementTracker Simulator::AllocateElement(const Element& e) {
	ElementTracker tracker;

	
	tracker.mass_begin = massBuf + numMasses;
	tracker.spring_begin = springBuf + numSprings;
	tracker.offset_begin = offsetBuf + numSprings;
	tracker.mass_end = tracker.mass_begin; 
	tracker.spring_end = tracker.spring_begin;
	tracker.offset_end = tracker.offset_begin;
	uint massOffset = numMasses;
	
	numElements++;
	
	for(const Mass& m : e.masses) {
		massBuf[numMasses] = m;
		tracker.mass_end++;
		numMasses++;
	}

	uint i = 0;

	unsigned seed = rand();
	std::vector<Spring> shuffledSprings(e.springs);
	std::shuffle(shuffledSprings.begin(), shuffledSprings.end(), std::default_random_engine(seed));

	for(const Spring& s : shuffledSprings) {
		springBuf[numSprings] = s;
		tracker.spring_end++;

		offsetBuf[numSprings] = massOffset;
		tracker.offset_end++;
		numSprings++;
		i++;
	}
	
	return tracker;
}

std::vector<Element> Simulator::Collect(const std::vector<ElementTracker>& trackers) {
	std::vector<Element> elements;
	for(const ElementTracker& tracker : trackers) {
		Element e = CollectElement(tracker);
		elements.push_back(e);
	}
	return elements;
}

Element Simulator::CollectElement(const ElementTracker& tracker) {
	std::vector<Mass> result_masses;
	std::vector<Spring> result_springs;

	for(Mass* i = tracker.mass_begin; i < tracker.mass_end; i++) {
		result_masses.push_back(*i);
	}

	for(Spring* i = tracker.spring_begin; i < tracker.spring_end; i++) {
		result_springs.push_back(*i);
	}
	
	return {result_masses, result_springs};
}
